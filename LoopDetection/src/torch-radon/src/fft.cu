#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include "utils.h"
#include "fft.h"
#include "log.h"

FFTConfig::FFTConfig(int dv, int r, int c):device(dv), rows(r), cols(c){}

bool FFTConfig::operator==(const FFTConfig &o) const{
    return device == o.device && rows == o.rows && cols == o.cols;
}

std::ostream &operator<<(std::ostream &os, FFTConfig const &cfg) {
    return os << "(device: " << cfg.device << ", rows: " << cfg.rows << ", cols: " << cfg.cols << ")";
}

FFTStructures::FFTStructures(const FFTConfig &_cfg) : cfg(_cfg) {
    LOG_INFO("Allocating FFT " << cfg);

    // create plans for FFT and iFFT
    cufftSafeCall(hipfftPlan1d(&forward_plan, cfg.cols, HIPFFT_R2C, cfg.rows));
    cufftSafeCall(hipfftPlan1d(&back_plan, cfg.cols, HIPFFT_C2R, cfg.rows));
}

bool FFTStructures::matches(const FFTConfig &k) const{
    return k == cfg;
}

FFTStructures::~FFTStructures() {
    // if (padded_data != nullptr) {
        LOG_DEBUG("Freeing FFT " << cfg);

        cufftSafeCall(hipfftDestroy(forward_plan));
        cufftSafeCall(hipfftDestroy(back_plan));
    // }
}


void FFT(FFTCache& fft_cache, const float *x, int device, int rows, int cols, float* y){
    FFTStructures* fft = fft_cache.get({device, rows, cols});
    checkCudaErrors(hipSetDevice(device));

    cufftSafeCall(hipfftExecR2C(fft->forward_plan, (hipfftReal *)x, (hipfftComplex *)y));
}

void iFFT(FFTCache& fft_cache, const float *x, int device, int rows, int cols, float* y){
    FFTStructures* fft = fft_cache.get({device, rows, cols});
    checkCudaErrors(hipSetDevice(device));

    cufftSafeCall(hipfftExecC2R(fft->back_plan, (hipfftComplex *)x, (hipfftReal *)y));
}